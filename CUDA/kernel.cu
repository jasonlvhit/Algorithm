#include "hip/hip_runtime.h"
#include <cstdio>
#include <fstream>
#include <iostream>
#include <time.h>
#include <windows.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

#include "cutil_inline.h"

#define numberOfVertex  500
#define Max_Iteration_Number 10000
#define Alpha 0.15
#define END_WEIGHT 1e-7
#define InitPageRankValue 6

using namespace std;
static int CPUiter = 0;


//PageRank value calculate function

__global__ void PRAdd(float *PR, const float* Graph, const float * sumOfOutDegree)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numberOfVertex )
    {
        float sum = 0.0;
        int k = 0;
        for (int j = i; j < numberOfVertex *numberOfVertex  ; j += numberOfVertex )
        {
            if (*(Graph + j) && sumOfOutDegree[k])
            {
                sum += PR[k] / sumOfOutDegree[k];

            }
            k++;
            //printf("%f\n", sum);
        } 
        PR[i] = Alpha  + (1 - Alpha)*sum;
    }
}

//Calculate Sum of out_degree of each vertex.
__global__ void claculateSumOfOutDegree(float * sumOfOutDegree, const float* Graph)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numberOfVertex )
    {
        sumOfOutDegree[i]  = 0;
        for (int j = 0; j < numberOfVertex ; ++j)
        {
            sumOfOutDegree[i] += *(Graph +i*numberOfVertex  +j);
        }
    }

}

//END condition: when the PR value stable

bool END(float a[], float b[])
{
    float sum = 0;
    for (int i = 0; i < numberOfVertex ; ++i)
    {
        sum += abs(a[i] - b[i]);
    }
    printf("The Deviation Between Two Iteration: %f \n",sum);
    if (sum < END_WEIGHT)
    {
        return true;
    }
    
    return false;
}

//CPU Routine of PageRank Calculation

int PageRank(float *Graph, float PR[])
{
    //Display the Graph:
/*
    for (int i = 0; i < vertex; ++i)
    {
        for (int j = 0; j < vertex; ++j)
        {
            printf("%f\t", *(Graph +i*vertex +j));
        }
        printf("\n");
    }
*/
    //Calculate the sum of out-degree of every vertex
    //eg. the sum of every line
    clock_t begin, end;
    float PR_Temp[numberOfVertex ];
    
    begin = clock();
    
    for (int m = 0; m < Max_Iteration_Number; ++m)
    {
        CPUiter++;
        float sumOfOutDegree[numberOfVertex ];
        for (int i = 0; i < numberOfVertex ; ++i)
        {
            sumOfOutDegree[i] = 0.0;
        }

        //Calculate the sum of degree of each vertex
        for (int i = 0; i < numberOfVertex ; ++i)
        {
            float sum = 0;
            for (int j = 0; j < numberOfVertex ; ++j)
            {
                sum += *(Graph +i*numberOfVertex  +j);
            }
            sumOfOutDegree[i] = sum;
        }

        //Calculate the PR value of every vertex.
        for (int i = 0; i < numberOfVertex ; ++i)
        {
            float sum = 0;
            int k = 0;
            for (int j = i; j < numberOfVertex *numberOfVertex  ; j += numberOfVertex )
            {
                if (*(Graph + j) == 1)
                {
                    if(sumOfOutDegree[k] != 0)
                        sum += PR[k] / sumOfOutDegree[k];
                }
                k++;
                //printf("%f\n", sum);
            }
            PR_Temp[i] = Alpha  + (1 - Alpha)*(sum);
        }

        if (END(PR_Temp, PR))
        {
            break;
        }
        else{
            for (int i = 0; i < numberOfVertex ; ++i)
            {
                PR[i] = PR_Temp[i];
            }
        }

    }
    end = clock();
    return end - begin;
}



int main()
{
    //char ch;
    int source = 0;
    int dest = 0;
    hipError_t err = hipSuccess;
    
    size_t size = numberOfVertex  * sizeof(float);

    float sumOfOutDegree[numberOfVertex ];

    //Allocate the device memory
    float *d_Sum_Of_Degree = NULL;
    hipMalloc((void **)&d_Sum_Of_Degree, size);
    if(d_Sum_Of_Degree == NULL)
    {
        cout << "Failed"<<endl;
    }

    float *d_PR = NULL;
    hipMalloc((void**)&d_PR,size);
    if (d_PR == NULL)
    {
        cout << "Failed" << endl;
    }

    float *d_Graph = NULL;
    
    hipMalloc((void **)&d_Graph, size * numberOfVertex );
    if (d_Graph == NULL)
    {
        cout <<"Failed" << endl;
    }

    //thread number

    int threadsPerBlock = numberOfVertex ;
    int blocksPerGrid =(numberOfVertex  + threadsPerBlock - 1) / threadsPerBlock;

    //Read Graph file.

    fstream fp("Graph.txt",ios::in);
    if(!fp.is_open())
    {
        printf("Failed to open file.\n");
    }

    //output file
    fstream prFile("PageRankValue.txt", ios::out);
    if (!prFile.is_open())
    {
        printf("Failed to open file PRV\n");
    }

    //host memory allocate

    float Graph[numberOfVertex ][numberOfVertex ];
    float PR[numberOfVertex ];
    float PR_Temp[numberOfVertex];


    //init
    for (int i = 0; i < numberOfVertex ; ++i)
    {
        PR[i] = InitPageRankValue;
        PR_Temp[i] = InitPageRankValue;
    }

    for (int i = 0; i < numberOfVertex ; ++i)
    {
        for (int j = 0; j < numberOfVertex ; ++j)
        {
            Graph[i][j] = 0;
        }
    }
	int edge = 0;
    //read from Graph.txt
    while (!fp.eof()){

        fp >> source >> dest;
        std::cout << source << ' '<< dest << std::endl;

        Graph[source][dest] = 1;
		edge++;
    }
    printf("Graph build Done!\n");
    printf("----------------------------------------------------------\n");


    //copy
    err = hipMemcpy(d_Graph, *Graph, numberOfVertex *size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //invoke PageRank.
	//CPU Routine
 
    int CPUTime  = 0;
    CPUTime = PageRank(*Graph, PR);


	for (int i = 0; i < numberOfVertex; ++i)
    {
        PR[i] = InitPageRankValue;
    }
    printf("--------------------------------------------------------\n");
    clock_t begin, end;
    int iter = 0;
    float SumOfGPUTime = 0;
    begin = clock();
    for (int m = 0; m < Max_Iteration_Number; ++m)
    {
        /*
        for (int i = 0; i < numberOfVertex ; ++i)
        {
            for (int j = 0; j < numberOfVertex ; ++j)
            {
                printf("%f\t", Graph[i][j]);
            }
            printf("\n");
        }
        */

        iter ++;

        //CUDA event timing
        
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        
        //calculate sum of out degree

        claculateSumOfOutDegree<<<blocksPerGrid, threadsPerBlock>>>(d_Sum_Of_Degree, d_Graph);
        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        /*
        hipMemcpy(sumOfOutDegree, d_Sum_Of_Degree, size, hipMemcpyDeviceToHost);
        for (int i = 0; i < numberOfVertex ; ++i)
        {
            cout << sumOfOutDegree[i] <<'\t';
        }
        */

        //copy
        err = hipMemcpy(d_PR, PR, size, hipMemcpyHostToDevice);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        PRAdd<<<blocksPerGrid, threadsPerBlock>>>(d_PR, d_Graph, d_Sum_Of_Degree);

        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);

        SumOfGPUTime += elapsedTime;

        hipMemcpy(PR_Temp, d_PR, size, hipMemcpyDeviceToHost);

        if (END(PR_Temp, PR))
        {
            break;
        }
        else{
            for (int i = 0; i < numberOfVertex ; ++i)
            {
                PR[i] = PR_Temp[i];
            }
        }
        

    }
    end = clock();
    


    //printf("%d\n", vertex);
    
    for (int i = 0; i < numberOfVertex; ++i)
    {
        prFile << i << "  :  " << PR[i] << endl;
    }
    

    


    printf("Matrix: %d * %d\n", numberOfVertex, numberOfVertex);
	printf("Edge: %d .\n", edge);
    printf("Number of thread : %d.\n", numberOfVertex);
    printf("Number of block : %d\n", blocksPerGrid);
    printf("--------------------------------------------------------\n");
    printf("CPU Routine of calculating %d iterations of PageRank value cost us:%d ms.\n",CPUiter,  CPUTime);
    printf("--------------------------------------------------------\n");

    printf("Calculation on GPU : %f ms.\n", SumOfGPUTime);
    printf("GPU Routine of calculating %d iterations of PageRank value cost us:%d ms.\n",iter,  end - begin);

    printf("--------------------------------\n");
    printf("Ratio of acceleration: %f \n", (float)SumOfGPUTime/CPUTime);
    printf("--------------------------------\n");

    fp.close();
    prFile.close();
    getchar();
}
